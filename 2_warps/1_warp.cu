#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include ""

/*
This is a one dimensional block. So threadIdx.x is sufficient. Otherwise you need to compute the id in 3D
threadId = threadIdx.x+blockDim.x*(threadIdx.y+blockDim.y*threadIdx.z). Every 32 threads of this index is a new warp.
*/
__global__ void print_details_of_warps()
{
	// global thread id
	int gid = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;

	// warp id: Each thread block is executed in warps (set of 32 threads. So local threadid/32)
	int warp_id = threadIdx.x / 32;

	// global block id
	int gbid = blockIdx.y * gridDim.x + blockIdx.x;

	std::cout << "local thread id = " << threadIdx.x << std::endl;
	std::cout << "block id in x = " << blockIdx.x << std::endl;
	std::cout << "block id in y = " << blockIdx.y << std::endl;
	std::cout << "block id in z = " << blockIdx.z << std::endl;
	std::cout << "global thread id = " << gid << std::endl;
	std::cout << "warp id = " << warp_id << std::endl;
	std::cout << "global block id = " << gbid << std::endl;
}

int main(int argc, char **argv)
{
	dim3 block_size(42);
	dim3 grid_size(2, 2);

	print_details_of_warps<<<grid_size, block_size>>>();
	hipDeviceSynchronize();

	hipDeviceReset();
	return EXIT_SUCCESS;
}