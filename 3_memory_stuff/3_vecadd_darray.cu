#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cassert>
#include <iostream>
#include "common_routines.h"
#include "d_array.h"

// Add vectors on the CPU
void vecadd_cpu(float*a, float* b, float* c, int N){
    for(int i=0; i<N; ++i){
        c[i] = a[i] + b[i];
    }
}

// Add vectors on the GPU
__global__ void vecadd_gpu(float *a, float *b, float *c, int N){
    
    int total_threads = blockDim.x * gridDim.x;
    int n_strides = int(N / total_threads) + 1;
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    for (int j=0; j<n_strides; ++j){
        if (i < N){
            c[i+j*n_strides] = a[i+j*n_strides] + b[i+j*n_strides];
        }
    }
}

int main(){

    int N = 10;
    size_t num_bytes = N * sizeof(float);

    // Host vectors: allocate
    float *a, *b, *c, *h_verify;

    a = (float*)malloc(num_bytes);
    b = (float*)malloc(num_bytes);
    c = (float*)malloc(num_bytes);
    h_verify = (float*)malloc(num_bytes);

    // Initialize vectors to random numbers
    init_random_vec(a, N, 1.0f, 10.0f);
    init_random_vec(b, N, 1.0f, 10.0f);  

    // Add the 2 vectors on CPU
    vecadd_cpu(a, b, c, N);

    // Device vectors
    d_array<float> d_a(N);
    d_array<float> d_b(N);
    d_array<float> d_c(N);

    // Copy vectors from host to device
    d_a.copy_host_to_device(a, N);
    d_b.copy_host_to_device(b, N);

    // Add vectors on GPU
    vecadd_gpu<<<1, 5>>>(d_a.getData(), d_b.getData(), d_c.getData(), N);
    hipDeviceSynchronize();

    // Copy the result vector from GPU to CPU
    d_c.copy_device_to_host(h_verify, N);

    //print_vector(c, N);
    //print_vector(h_verify, N);

    // Check if the CPU result is the same as the GPU result
    check_equality(c, h_verify, N);

    // Free the resources
    free(a);
    free(b);
    free(c);
    free(h_verify);

    return 0;
}