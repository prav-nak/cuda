#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cassert>
#include <iostream>
#include "common_routines.h"

// Add vectors on the CPU
void vecadd_cpu(float*a, float* b, float* c, int N){
    for(int i=0; i<N; ++i){
        c[i] = a[i] + b[i];
    }
}

// Add vectors on the GPU
__global__ void vecadd_gpu(float *a, float *b, float *c, int N){
    
    int total_threads = blockDim.x * gridDim.x;
    int n_strides = int(N / total_threads) + 1;
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    for (int j=0; j<n_strides; ++j){
        if (i < N){
            c[i+j*n_strides] = a[i+j*n_strides] + b[i+j*n_strides];
        }
    }
}

int main(){

    int N = 10;
    size_t num_bytes = N * sizeof(float);

    // Host vectors: allocate
    float *a, *b, *c, *h_verify;

    a = (float*)malloc(num_bytes);
    b = (float*)malloc(num_bytes);
    c = (float*)malloc(num_bytes);
    h_verify = (float*)malloc(num_bytes);

    // Initialize vectors to random numbers
    init_random_vec(a, N, 1.0f, 10.0f);
    init_random_vec(b, N, 1.0f, 10.0f);  

    // Add the 2 vectors on CPU
    vecadd_cpu(a, b, c, N);

    // Device vectors
    float *d_a;
    float *d_b;
    float *d_c;
    hipMalloc((void**)&d_a, num_bytes);
    hipMalloc((void**)&d_b, num_bytes);
    hipMalloc((void**)&d_c, num_bytes);

    // Copy vectors from host to device
    hipMemcpy(d_a, a, num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, num_bytes, hipMemcpyHostToDevice);

    // Add vectors on GPU
    vecadd_gpu<<<1, 5>>>(d_a, d_b, d_c, N);
    hipDeviceSynchronize();

    // Copy the result vector from GPU to CPU
    hipMemcpy(h_verify, d_c, num_bytes, hipMemcpyDeviceToHost);

    //print_vector(c, N);
    //print_vector(h_verify, N);

    // Check if the CPU result is the same as the GPU result
    check_equality(c, h_verify, N);

    // Free the resources
    free(a);
    free(b);
    free(c);
    free(h_verify);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}