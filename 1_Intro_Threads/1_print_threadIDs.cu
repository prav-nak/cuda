#include "hip/hip_runtime.h"


#include <stdio.h>


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// Function to print the ID of the threads
__global__ void print_threadIds()
{
	printf("threadIdx.x: %d, threadIdx.y: %d, threadIdx.z:%d\n", threadIdx.x, threadIdx.y, threadIdx.z);
}

int main()
{
	int nx, ny, nz;

	nx = 2;
	ny = 3;
	nz = 1;

	dim3 grid_size(1, 1, 1);
	dim3 block_size(nx, ny, nz);

	print_threadIds<<<grid_size, block_size>>>();
	gpuErrchk( hipPeekAtLastError() );

	hipDeviceSynchronize();
	hipDeviceReset();
	return 0;
}
