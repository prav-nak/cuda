#include "hip/hip_runtime.h"


#include <stdio.h>

// Function to print the ID of the threads
__global__ void print_threadIds()
{
	printf("threadIdx.x: %d, threadIdx.y: %d, threadIdx.z:%d\n", threadIdx.x, threadIdx.y, threadIdx.z);
}

int main()
{
	int nx, ny, nz;

	nx = 2;
	ny = 3;
	nz = 1;

	dim3 grid_size(1, 1, 1);
	dim3 block_size(nx, ny, nz);

	print_threadIds<<<grid_size, block_size>>>();

	hipDeviceSynchronize();
	hipDeviceReset();
	return 0;
}
