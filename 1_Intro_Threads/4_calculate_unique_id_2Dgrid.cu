#include "hip/hip_runtime.h"


#include <stdio.h>

// For a 2D grid, print the unique ID of the thread
__global__ void unique_globalid_2Darray_calc(int *input)
{
    int nrows_to_skip = blockIdx.y;
    int nthreads_in_each_block = blockDim.x * blockDim.y;
    int n_ind_for_skipped_rows = nrows_to_skip * nthreads_in_each_block * gridDim.y;
    
    int nblocks_to_skip_eachrow = blockIdx.x;
    int n_ind_for_skipped_blocks = nblocks_to_skip_eachrow * nthreads_in_each_block;

    int local_threadid = threadIdx.y * blockDim.x + threadIdx.x;
    int gid = n_ind_for_skipped_rows + n_ind_for_skipped_blocks + local_threadid;

    printf("nrows_to_skip: %d, n_ind_for_skipped_rows: %d, nblocks_to_skip_eachrow: %d, n_ind_for_skipped_blocks: %d, local_threadid: %d, gid: %d\n", nrows_to_skip, n_ind_for_skipped_rows, nblocks_to_skip_eachrow, n_ind_for_skipped_blocks, local_threadid, gid);
    printf("value of the array at index %d is %d\n", gid, input[gid]);
}

int main()
{
    int array_size = 16;

    int array_byte_size = sizeof(int) * array_size;

    // Create the host data array
    int host_data[array_size];
    for (int i = 0; i < array_size; ++i)
        host_data[i] = i;

    // Create the device data array
    int *device_data;
    hipMalloc((void **)&device_data, array_byte_size);
    hipMemcpy(device_data, host_data, array_byte_size, hipMemcpyHostToDevice);

    dim3 grid_size(2, 2, 1);
    dim3 block_size(2, 2, 1);

    // Get the array components using device
    unique_globalid_2Darray_calc<<<grid_size, block_size>>>(device_data);

    hipFree(device_data);

    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}
