
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>

/*-----------------------------------;
//Print the properties of the device ;
-------------------------------------;*/
void query_device()
{
	int iDev = 0;
	hipDeviceProp_t iProp;
	hipGetDeviceProperties(&iProp, iDev);

	printf("Device %d: %s\n", iDev, iProp.name);
	printf("  Number of multiprocessors:                     %d\n", iProp.multiProcessorCount);
	printf("  Compute capability       :                     %d.%d\n", iProp.major,iProp.minor);
	printf("  Total amount of global memory:                 %4.2f KB\n", iProp.totalGlobalMem/ 1024.0);
	printf("  Total amount of constant memory:               %4.2f KB\n", iProp.totalConstMem / 1024.0);
	printf("  Total amount of shared memory per block:       %4.2f KB\n", iProp.sharedMemPerBlock / 1024.0);
	printf("  Total amount of shared memory per MP:          %4.2f KB\n", iProp.sharedMemPerMultiprocessor / 1024.0);
	printf("  Total number of registers available per block: %d\n", iProp.regsPerBlock);
	printf("  Warp size:                                     %d\n", iProp.warpSize);
	printf("  Maximum number of threads per block:           %d\n", iProp.maxThreadsPerBlock);
	printf("  Maximum number of threads per multiprocessor:  %d\n", iProp.maxThreadsPerMultiProcessor);
	printf("  Maximum number of warps per multiprocessor:    %d\n", iProp.maxThreadsPerMultiProcessor / 32);
	printf("  Maximum Grid size                         :    (%d,%d,%d)\n", iProp.maxGridSize[0], iProp.maxGridSize[1], iProp.maxGridSize[2]);
	printf("  Maximum block dimension                   :    (%d,%d,%d)\n", iProp.maxThreadsDim[0], iProp.maxThreadsDim[1], iProp.maxThreadsDim[2]);
}

int main(){
    query_device();
}
